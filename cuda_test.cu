#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

// Kernel function to add the elements of two arrays
__global__ void addNums(int *output, int *x, int *y, int num_iters) {
  for (int i = 0; i < num_iters; i++) {
    output[i] = x[i] + y[i];
  }
}

int main() {
  // Declare the variables
  int num_iters = 12000000;
  int *x;
  int *y;
  int *output;
  
  // Seeding the random number generator
  srand(10);

  std::cout << "Hello World, this is CUDA sample code" << std::endl;
  
  // Declare the memory size for the variables from the unified memory location accessible from CPU or GPU
  hipMallocManaged(&x, num_iters*sizeof(int));
  hipMallocManaged(&y, num_iters*sizeof(int));
  hipMallocManaged(&output, num_iters*sizeof(int));

  // Initialization with random numbers
  for (unsigned int i = 0; i < num_iters; i++) {
    output[i] = 0;
    x[i] = rand();
    y[i] = rand();
  }

  // Run the kernel function on each 64 threads on 32 blocks of the GPU
  addNums<<<32, 64>>>(output, x, y, num_iters);

  // Synchronization between the CPU and GPU (CPU waiting for GPU to finish before accessing the memory)
  hipDeviceSynchronize();

  // Releasing the memory
  hipFree(x);
  hipFree(y);

  std::cout << "Code Execution Completed" << std::endl;  

  return 0;
}
